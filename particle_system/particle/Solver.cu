#include "hip/hip_runtime.h"
#include "Solver.h"

#include <stdio.h>
#include <stdlib.h>

#include <perf.h>
#include "constants.h"
#include "common/debug.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define checkCudaErrors(err) err;

void genLaplace(int *row_ptr, int *col_ind, float *val, int M, int N, int nz,
                float *rhs)
{
    assert(M == N);
    int n = (int)sqrt((double)N);
    assert(n * n == N);
    printf("laplace dimension = %d\n", n);
    int idx = 0;

    // loop over degrees of freedom
    for (int i = 0; i < N; i++)
    {
        int ix = i % n;
        int iy = i / n;

        row_ptr[i] = idx;

        // up
        if (iy > 0)
        {
            val[idx] = 1.0;
            col_ind[idx] = i - n;
            idx++;
        }
        else
        {
            rhs[i] -= 1.0;
        }

        // left
        if (ix > 0)
        {
            val[idx] = 1.0;
            col_ind[idx] = i - 1;
            idx++;
        }
        else
        {
            rhs[i] -= 0.0;
        }

        // center
        val[idx] = -4.0;
        col_ind[idx] = i;
        idx++;

        // right
        if (ix < n - 1)
        {
            val[idx] = 1.0;
            col_ind[idx] = i + 1;
            idx++;
        }
        else
        {
            rhs[i] -= 0.0;
        }

        // down
        if (iy < n - 1)
        {
            val[idx] = 1.0;
            col_ind[idx] = i + n;
            idx++;
        }
        else
        {
            rhs[i] -= 0.0;
        }
    }

    row_ptr[N] = idx;
}

void cudaSolve(
    Eigen::SparseMatrix<float, Eigen::RowMajor> &At,
    Eigen::VectorXf &bt,
    Eigen::VectorXf &xt)
{
    //--------------------------------------------------------------------------

    const int max_iter = 1000;
    int k, M = 0, N = 0, nz = 0, *I = NULL, *J = NULL;
    int *d_col, *d_row;
    int qatest = 0;
    const float tol = 1e-5f;
    float *x, *rhs;
    float r0, r1, alpha, beta;
    float *d_val, *d_x;
    float *d_zm1, *d_zm2, *d_rm2;
    float *d_r, *d_p, *d_omega, *d_y;
    float *val = NULL;
    float *d_valsILU0;
    float rsum, diff, err = 0.0;
    float qaerr1, qaerr2 = 0.0;
    float dot, numerator, denominator, nalpha;
    const float floatone = 1.0;
    const float floatzero = 0.0;

    int nErrors = 0;

    printf("Nz = %d\n", At.nonZeros());
    
    //  I : row pointer, J : column pointer, val : value pointer,
    //  N : number of rows, nz : number of non-zero elements

    // N = At.rows();
    // nz = At.nonZeros();
    // I = At.outerIndexPtr();
    // J = At.innerIndexPtr();
    // val = At.valuePtr();

    // x = xt.data();
    // rhs = bt.data();
    // printf("rhs size = %d\n", bt.size());
    // printf("x size = %d\n", xt.size());

    /* Generate a Laplace matrix in CSR (Compressed Sparse Row) format */

    M = N = 32*64*32;
    nz = 5 * N - 4 * (int)sqrt((double)N);
    I = (int *)malloc(sizeof(int) * (N + 1));   // csr row pointers for matrix A
    J = (int *)malloc(sizeof(int) * nz);       // csr column indices for matrix A
    val = (float *)malloc(sizeof(float) * nz); // csr values for matrix A
    x = (float *)malloc(sizeof(float) * N);
    rhs = (float *)malloc(sizeof(float) * N);    


    for (int i = 0; i < N; i++)
    {
        rhs[i] = 0.0;  // Initialize RHS
        x[i] = 0.0;    // Initial solution approximation
    }

    genLaplace(I, J, val, M, N, nz, rhs);

    printf("nz = %d\n", nz);

    /* Create CUBLAS context */
    hipblasHandle_t cublasHandle = NULL;
    checkCudaErrors(hipblasCreate(&cublasHandle));

    /* Create CUSPARSE context */
    hipsparseHandle_t cusparseHandle = NULL;
    checkCudaErrors(hipsparseCreate(&cusparseHandle));

    /* Description of the A matrix */
    hipsparseMatDescr_t descr = 0;
    checkCudaErrors(hipsparseCreateMatDescr(&descr));
    checkCudaErrors(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    /* Allocate required memory */
    checkCudaErrors(hipMalloc((void **)&d_col, nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_row, (N + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_val, nz * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_x, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_y, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_p, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_omega, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_valsILU0, nz * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_zm1, (N) * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_zm2, (N) * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_rm2, (N) * sizeof(float)));

    /* Wrap raw data into cuSPARSE generic API objects */
    hipsparseDnVecDescr_t vecp = NULL, vecX = NULL, vecY = NULL, vecR = NULL, vecZM1 = NULL;
    checkCudaErrors(hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecX, N, d_x, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecY, N, d_y, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecR, N, d_r, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecZM1, N, d_zm1, HIP_R_32F));
    hipsparseDnVecDescr_t vecomega = NULL;
    checkCudaErrors(hipsparseCreateDnVec(&vecomega, N, d_omega, HIP_R_32F));

    /* Initialize problem data */
    checkCudaErrors(hipMemcpy(
        d_col, J, nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        d_row, I, (N + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        d_val, val, nz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        d_val, val, nz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        d_x, x, N * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        d_r, rhs, N * sizeof(float), hipMemcpyHostToDevice));

    hipsparseSpMatDescr_t matA = NULL;
    hipsparseSpMatDescr_t matM_lower, matM_upper;
    hipsparseFillMode_t fill_lower = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t diag_unit = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t fill_upper = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

    checkCudaErrors(hipsparseCreateCsr(
        &matA, N, N, nz, d_row, d_col, d_val, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    /* Copy A data to ILU(0) vals as input*/
    checkCudaErrors(hipMemcpy(
        d_valsILU0, d_val, nz * sizeof(float), hipMemcpyDeviceToDevice));

    // Lower Part
    checkCudaErrors(hipsparseCreateCsr(&matM_lower, N, N, nz, d_row, d_col, d_valsILU0,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    checkCudaErrors(hipsparseSpMatSetAttribute(matM_lower,
                                              HIPSPARSE_SPMAT_FILL_MODE,
                                              &fill_lower, sizeof(fill_lower)));
    checkCudaErrors(hipsparseSpMatSetAttribute(matM_lower,
                                              HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diag_unit, sizeof(diag_unit)));
    // M_upper
    checkCudaErrors(hipsparseCreateCsr(&matM_upper, N, N, nz, d_row, d_col, d_valsILU0,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    checkCudaErrors(hipsparseSpMatSetAttribute(matM_upper,
                                              HIPSPARSE_SPMAT_FILL_MODE,
                                              &fill_upper, sizeof(fill_upper)));
    checkCudaErrors(hipsparseSpMatSetAttribute(matM_upper,
                                              HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diag_non_unit,
                                              sizeof(diag_non_unit)));

    /* Create ILU(0) info object */
    int bufferSizeLU = 0;
    size_t bufferSizeMV, bufferSizeL, bufferSizeU;
    void *d_bufferLU, *d_bufferMV, *d_bufferL, *d_bufferU;
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseMatDescr_t matLU;
    csrilu02Info_t infoILU = NULL;

    checkCudaErrors(hipsparseCreateCsrilu02Info(&infoILU));
    checkCudaErrors(hipsparseCreateMatDescr(&matLU));
    checkCudaErrors(hipsparseSetMatType(matLU, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(matLU, HIPSPARSE_INDEX_BASE_ZERO));

    /* Allocate workspace for cuSPARSE */
    checkCudaErrors(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA,
        vecp, &floatzero, vecomega, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT,
        &bufferSizeMV));
    checkCudaErrors(hipMalloc(&d_bufferMV, bufferSizeMV));

    checkCudaErrors(hipsparseScsrilu02_bufferSize(
        cusparseHandle, N, nz, matLU, d_val, d_row, d_col, infoILU, &bufferSizeLU));
    checkCudaErrors(hipMalloc(&d_bufferLU, bufferSizeLU));

    checkCudaErrors(hipsparseSpSV_createDescr(&spsvDescrL));
    checkCudaErrors(hipsparseSpSV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matM_lower, vecR, vecX, HIP_R_32F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &bufferSizeL));
    checkCudaErrors(hipMalloc(&d_bufferL, bufferSizeL));

    checkCudaErrors(hipsparseSpSV_createDescr(&spsvDescrU));
    checkCudaErrors(hipsparseSpSV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matM_upper, vecR, vecX, HIP_R_32F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &bufferSizeU));
    checkCudaErrors(hipMalloc(&d_bufferU, bufferSizeU));

    /* Preconditioned Conjugate Gradient using ILU.
       --------------------------------------------
       Follows the description by Golub & Van Loan,
       "Matrix Computations 3rd ed.", Algorithm 10.3.1  */

    // printf("\nConvergence of CG using ILU(0) preconditioning: \n");

    /* Perform analysis for ILU(0) */
    checkCudaErrors(hipsparseScsrilu02_analysis(
        cusparseHandle, N, nz, descr, d_valsILU0, d_row, d_col, infoILU,
        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_bufferLU));

    /* generate the ILU(0) factors */
    checkCudaErrors(hipsparseScsrilu02(
        cusparseHandle, N, nz, matLU, d_valsILU0, d_row, d_col, infoILU,
        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_bufferLU));

    /* perform triangular solve analysis */
    checkCudaErrors(hipsparseSpSV_analysis(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone,
        matM_lower, vecR, vecX, HIP_R_32F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, d_bufferL));

    checkCudaErrors(hipsparseSpSV_analysis(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone,
        matM_upper, vecR, vecX, HIP_R_32F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, d_bufferU));

    /* reset the initial guess of the solution to zero */
    for (int i = 0; i < N; i++)
    {
        x[i] = 0.0;
    }
    checkCudaErrors(hipMemcpy(
        d_r, rhs, N * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        d_x, x, N * sizeof(float), hipMemcpyHostToDevice));

    k = 0;
    checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1));

    while (r1 > tol * tol && k <= max_iter)
    {
        // preconditioner application: d_zm1 = U^-1 L^-1 d_r
        checkCudaErrors(hipsparseSpSV_solve(cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone,
                                           matM_lower, vecR, vecY, HIP_R_32F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescrL));

        checkCudaErrors(hipsparseSpSV_solve(cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matM_upper,
                                           vecY, vecZM1,
                                           HIP_R_32F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescrU));
        k++;

        if (k == 1)
        {
            checkCudaErrors(hipblasScopy(cublasHandle, N, d_zm1, 1, d_p, 1));
        }
        else
        {
            checkCudaErrors(hipblasSdot(
                cublasHandle, N, d_r, 1, d_zm1, 1, &numerator));
            checkCudaErrors(hipblasSdot(
                cublasHandle, N, d_rm2, 1, d_zm2, 1, &denominator));
            beta = numerator / denominator;
            checkCudaErrors(hipblasSscal(cublasHandle, N, &beta, d_p, 1));
            checkCudaErrors(hipblasSaxpy(
                cublasHandle, N, &floatone, d_zm1, 1, d_p, 1));
        }

        checkCudaErrors(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA,
            vecp, &floatzero, vecomega, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT,
            d_bufferMV));
        checkCudaErrors(hipblasSdot(
            cublasHandle, N, d_r, 1, d_zm1, 1, &numerator));
        checkCudaErrors(hipblasSdot(
            cublasHandle, N, d_p, 1, d_omega, 1, &denominator));
        alpha = numerator / denominator;
        checkCudaErrors(hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1));
        checkCudaErrors(hipblasScopy(cublasHandle, N, d_r, 1, d_rm2, 1));
        checkCudaErrors(hipblasScopy(cublasHandle, N, d_zm1, 1, d_zm2, 1));
        nalpha = -alpha;
        checkCudaErrors(hipblasSaxpy(
            cublasHandle, N, &nalpha, d_omega, 1, d_r, 1));
        checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1));
    }

    printf("  iteration = %3d, residual = %e \n", k, sqrt(r1));

    checkCudaErrors(hipMemcpy(
        x, d_x, N * sizeof(float), hipMemcpyDeviceToHost));

    /* Destroy descriptors */
    checkCudaErrors(hipsparseDestroyCsrilu02Info(infoILU));
    checkCudaErrors(hipsparseDestroyMatDescr(matLU));
    checkCudaErrors(hipsparseSpSV_destroyDescr(spsvDescrL));
    checkCudaErrors(hipsparseSpSV_destroyDescr(spsvDescrU));
    checkCudaErrors(hipsparseDestroySpMat(matM_lower));
    checkCudaErrors(hipsparseDestroySpMat(matM_upper));
    checkCudaErrors(hipsparseDestroySpMat(matA));
    checkCudaErrors(hipsparseDestroyDnVec(vecp));
    checkCudaErrors(hipsparseDestroyDnVec(vecomega));
    checkCudaErrors(hipsparseDestroyDnVec(vecR));
    checkCudaErrors(hipsparseDestroyDnVec(vecX));
    checkCudaErrors(hipsparseDestroyDnVec(vecY));
    checkCudaErrors(hipsparseDestroyDnVec(vecZM1));

    /* Destroy contexts */
    checkCudaErrors(hipsparseDestroy(cusparseHandle));
    checkCudaErrors(hipblasDestroy(cublasHandle));

    /* Free device memory */
    free(I);
    free(J);
    free(val);
    free(x);
    free(rhs);
    checkCudaErrors(hipFree(d_bufferMV));
    checkCudaErrors(hipFree(d_bufferLU));
    checkCudaErrors(hipFree(d_bufferL));
    checkCudaErrors(hipFree(d_bufferU));
    checkCudaErrors(hipFree(d_col));
    checkCudaErrors(hipFree(d_row));
    checkCudaErrors(hipFree(d_val));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_r));
    checkCudaErrors(hipFree(d_p));
    checkCudaErrors(hipFree(d_omega));
    checkCudaErrors(hipFree(d_valsILU0));
    checkCudaErrors(hipFree(d_zm1));
    checkCudaErrors(hipFree(d_zm2));
    checkCudaErrors(hipFree(d_rm2));

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
}