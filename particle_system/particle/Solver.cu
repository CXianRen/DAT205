#include "Solver.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>


void cudaSolve()
{
    // problem definition
    const int A_num_rows = 40;
    const int A_num_cols = 40;
    const int A_nnz = 80; // assuming a dense matrix
    int hA_csrOffsets[A_num_rows + 1];
    int hA_columns[A_nnz];
    float hA_values[A_nnz];
    float hX[A_num_cols];
    float hY[A_num_rows];
    float hY_result[A_num_rows];
    float alpha = 1.0f;
    float beta = 0.0f;

    // Initialize your data here. For example:
    for (int i = 0; i < A_num_rows + 1; i++)
    {
        hA_csrOffsets[i] = i * A_num_cols; // assuming a dense matrix
    }
    for (int i = 0; i < A_nnz; i++)
    {
        hA_columns[i] = i % A_num_cols;                // assuming a dense matrix
        hA_values[i] = static_cast<float>(i % 10 + 1); // some arbitrary values
    }
    for (int i = 0; i < A_num_cols; i++)
    {
        hX[i] = static_cast<float>(i % 10 + 1); // some arbitrary values
    }
    for (int i = 0; i < A_num_rows; i++)
    {
        hY[i] = 0.0f;
        hY_result[i] = 0.0f; // you'll need to compute the correct results
    }

    //--------------------------------------------------------------------------
    // Device memory management
    int *dA_csrOffsets, *dA_columns;
    float *dA_values, *dX, *dY;
    hipMalloc((void **)&dA_csrOffsets, (A_num_rows + 1) * sizeof(int));
    hipMalloc((void **)&dA_columns, A_nnz * sizeof(int));
    hipMalloc((void **)&dA_values, A_nnz * sizeof(float));
    hipMalloc((void **)&dX, A_num_cols * sizeof(float));
    hipMalloc((void **)&dY, A_num_rows * sizeof(float));

    hipMemcpy(dA_csrOffsets, hA_csrOffsets, (A_num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dX, hX, A_num_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dY, hY, A_num_rows * sizeof(float), hipMemcpyHostToDevice);
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    hipsparseCreate(&handle);
    hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                      dA_csrOffsets, dA_columns, dA_values,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_32F);
    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute SpMV
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);

    // destroy matrix/vector descriptors
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);
    //--------------------------------------------------------------------------
    // device result check
    hipMemcpy(hY, dY, A_num_rows * sizeof(float), hipMemcpyDeviceToHost);
    int correct = 1;
    for (int i = 0; i < A_num_rows; i++)
    {
        if (hY[i] != hY_result[i])
        {                // direct floating point comparison is not
            correct = 0; // reliable
            break;
        }
    }
    if (correct)
        printf("spmv_csr_example test PASSED\n");
    else
        printf("spmv_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    hipFree(dBuffer);
    hipFree(dA_csrOffsets);
    hipFree(dA_columns);
    hipFree(dA_values);
    hipFree(dX);
    hipFree(dY);
}