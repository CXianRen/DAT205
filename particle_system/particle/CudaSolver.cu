#include "hip/hip_runtime.h"
#include "Solver.h"

#include <stdio.h>
#include <stdlib.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define checkCudaErrors(err)                                         \
    if (err != hipSuccess)                                          \
    {                                                                \
        printf("CUDA error\n");                                      \
        printf("Error at line %d in file %s\n", __LINE__, __FILE__); \
        exit(1);                                                     \
    }

void __genLaplace(int *row_ptr, int *col_ind, float *val, int M, int N, int nz,
                  float *rhs)
{
    assert(M == N);
    int n = (int)sqrt((double)N);
    assert(n * n == N);
    printf("laplace dimension = %d\n", n);
    int idx = 0;

    // loop over degrees of freedom
    for (int i = 0; i < N; i++)
    {
        int ix = i % n;
        int iy = i / n;

        row_ptr[i] = idx;

        // up
        if (iy > 0)
        {
            val[idx] = 1.0;
            col_ind[idx] = i - n;
            idx++;
        }
        else
        {
            rhs[i] -= 1.0;
        }

        // left
        if (ix > 0)
        {
            val[idx] = 1.0;
            col_ind[idx] = i - 1;
            idx++;
        }
        else
        {
            rhs[i] -= 0.0;
        }

        // center
        val[idx] = -4.0;
        col_ind[idx] = i;
        idx++;

        // right
        if (ix < n - 1)
        {
            val[idx] = 1.0;
            col_ind[idx] = i + 1;
            idx++;
        }
        else
        {
            rhs[i] -= 0.0;
        }

        // down
        if (iy < n - 1)
        {
            val[idx] = 1.0;
            col_ind[idx] = i + n;
            idx++;
        }
        else
        {
            rhs[i] -= 0.0;
        }
    }

    row_ptr[N] = idx;
}

void CudaSolver::compute(
    Eigen::SparseMatrix<double, Eigen::RowMajor> &A)
{
    N = A.rows();
    nz = A.nonZeros();
    I = A.outerIndexPtr();
    J = A.innerIndexPtr();
    val = A.valuePtr();

    hipblasStatus_t = hipblasCreate(&cublasHandle);
    checkCudaErrors(hipblasStatus_t);

    /* Get handle to the CUSPARSE context */
    checkCudaErrors(hipsparseCreate(&cusparseHandle));

    checkCudaErrors(hipMalloc((void **)&d_col, nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_row, (N + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_val, nz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_x, N * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_r, N * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_p, N * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_Ax, N * sizeof(double)));

    /* Wrap raw data into cuSPARSE generic API objects */
    checkCudaErrors(hipsparseCreateCsr(&matA, N, N, nz, d_row, d_col, d_val,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    checkCudaErrors(hipsparseCreateDnVec(&vecx, N, d_x, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecAx, N, d_Ax, HIP_R_64F));

    /* Initialize problem data */
    hipMemcpy(d_col, J, nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz * sizeof(double), hipMemcpyHostToDevice);
}

void CudaSolver::getIterations(int &iter)
{
    iter = k;
}

void CudaSolver::getError(double &error)
{
    double rsum, diff, err = 0.0;

    for (int i = 0; i < N; i++)
    {
        rsum = 0.0;

        for (int j = I[i]; j < I[i + 1]; j++)
        {
            rsum += val[j] * x[J[j]];
        }

        diff = fabs(rsum - rhs[i]);

        if (diff > err)
        {
            err = diff;
        }
    }
    error = err;
}

void CudaSolver::solve(
    Eigen::VectorXd &xt,
    Eigen::VectorXd &bt)
{
    x = xt.data();
    rhs = bt.data();
    /* Initialize problem data */
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_r, rhs, N * sizeof(double), hipMemcpyHostToDevice);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.;

    /* Allocate workspace for cuSPARSE */
    static void *buffer = NULL;
    {
        static bool do_one = true;
        if (do_one)
        {
            do_one = false;
            size_t bufferSize = 0;
            checkCudaErrors(hipsparseSpMV_bufferSize(
                cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
                &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));

            checkCudaErrors(hipMalloc(&buffer, bufferSize));
        }
    }

    /* Begin CG */
    checkCudaErrors(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecx, &beta, vecAx, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    hipblasDaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
    hipblasStatus_t = hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);

    k = 1;
    // printf("tol * tol = %e\n", tol * tol);
    auto lowest = sqrt(r1);
    while (r1 > tol * tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
            hipblasStatus_t = hipblasDscal(cublasHandle, N, &b, d_p, 1);
            hipblasStatus_t = hipblasDaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
        }
        else
        {
            hipblasStatus_t = hipblasDcopy(cublasHandle, N, d_r, 1, d_p, 1);
        }

        checkCudaErrors(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp,
            &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
        hipblasStatus_t = hipblasDdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
        a = r1 / dot;

        hipblasStatus_t = hipblasDaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);
        na = -a;
        hipblasStatus_t = hipblasDaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);

        r0 = r1;
        hipblasStatus_t = hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        hipDeviceSynchronize();
        // printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        if (sqrt(r1) < lowest)
        {
            lowest = sqrt(r1);
        }

        k++;
    }
    
    // printf("lowest error = %e\n", lowest);
    // printf("iteration = %3d, residual = %e\n", k, sqrt(r1));

    // copy result back to host
    hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);
}

CudaSolver::~CudaSolver()
{
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    if (matA)
    {
        checkCudaErrors(hipsparseDestroySpMat(matA));
    }
    if (vecx)
    {
        checkCudaErrors(hipsparseDestroyDnVec(vecx));
    }
    if (vecAx)
    {
        checkCudaErrors(hipsparseDestroyDnVec(vecAx));
    }
    if (vecp)
    {
        checkCudaErrors(hipsparseDestroyDnVec(vecp));
    }

    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);

    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);
}