#include "hip/hip_runtime.h"
#include "CudaWorker.h"

#include <hip/hip_runtime.h>

#include "common/debug.h"
#include "common/mmath.h"
#include "SimBase.h"

#define CUDA_FOR_EACH                                \
    int idx = blockIdx.x * blockDim.x + threadIdx.x; \
    /*calculate the index of the cell*/              \
    int i = idx % Nx;                                \
    int j = (idx / Nx) % Ny;                         \
    int k = idx / (Nx * Ny);

namespace MCUDA
{
    __global__ void calculateAverageVelocityKernel(
        double *u, double *v, double *w,
        double *avg_u, double *avg_v, double *avg_w,
        int workSize, int Nx, int Ny, int Nz)
    {
        CUDA_FOR_EACH;
        if (idx < workSize)
        {
            calculateAverageVelocity(
                i, j, k,
                Nx, Ny, Nz,
                u, v, w,
                avg_u, avg_v, avg_w);
        }
    }

    __global__ void calculateOmgKernel(
        double *avg_u, double *avg_v, double *avg_w,
        double *omg_x, double *omg_y, double *omg_z,
        int workSize, int Nx, int Ny, int Nz)
    {
        CUDA_FOR_EACH
        if (idx < workSize)
        {
            calculateGradient(
                i, j, k,
                Nx, Ny, Nz,
                avg_u, avg_v, avg_w,
                omg_x, omg_y, omg_z);
        }
    }

    __global__ void calculateVorticityForceKernel(
        double *omg_x, double *omg_y, double *omg_z,
        double *f_x, double *f_y, double *f_z,
        int workSize, int Nx, int Ny, int Nz)
    {

        CUDA_FOR_EACH
        if (idx < workSize)
        {
            calculateVorticityBody<double>(
                i, j, k,
                Nx, Ny, Nz,
                omg_x, omg_y, omg_z,
                f_x, f_y, f_z);
        }
    }

    __global__ void applyExternalForceKernel(
        double *u, double *v, double *w,
        double *f_x, double *f_y, double *f_z,
        int workSize, int Nx, int Ny, int Nz)
    {
        CUDA_FOR_EACH
        if (idx < workSize)
        {
            applyExternalForceBody(
                i, j, k,
                Nx, Ny, Nz,
                f_x, f_y, f_z,
                u, v, w);
        }
    }

    __global__ void advectVelocityFieldKernel(
        double *u, double *v, double *w,
        double *u_0, double *v_0, double *w_0,
        int workSize, int Nx, int Ny, int Nz)
    {
        CUDA_FOR_EACH
        if (idx < workSize)
        {
            advectVelocityBody<double>(
                u, v, w,
                u_0, v_0, w_0,
                i, j, k,
                Nx, Ny, Nz);
        }
    }

    __global__ void advectScalarFieldKernel(
        double *field, double *field_0,
        double *u_0, double *v_0, double *w_0,
        int workSize, int Nx, int Ny, int Nz)
    {
        CUDA_FOR_EACH
        if (idx < workSize)
        {
            advectScalarBody<double>(
                i, j, k,
                Nx, Ny, Nz,
                field, field_0,
                u_0, v_0, w_0);
        }
    }

    CudaWorker::CudaWorker(
        int workSize,
        int Nx,
        int Ny,
        int Nz) : workSize_(workSize), Nx_(Nx), Ny_(Ny), Nz_(Nz)
    {
        // check cuda device properties
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        if (deviceCount == 0)
        {
            DEBUG_PRINT("No CUDA devices found");
            exit(1);
        }
        DEBUG_PRINT("CUDA Device Count: " << deviceCount);
        hipSetDevice(0);

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        DEBUG_PRINT("CUDA Device Name: " << deviceProp.name);
        DEBUG_PRINT("CUDA Compute Capability: " << deviceProp.major << "." << deviceProp.minor);
        DEBUG_PRINT("CUDA Device Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << "MB");

        // SM count
        int SMCount = deviceProp.multiProcessorCount;
        DEBUG_PRINT("CUDA SM Count: " << SMCount);

        // max grid size
        int maxGridSize[3];
        maxGridSize[0] = deviceProp.maxGridSize[0];
        maxGridSize[1] = deviceProp.maxGridSize[1];
        maxGridSize[2] = deviceProp.maxGridSize[2];
        DEBUG_PRINT("CUDA Max Grid Size: " << maxGridSize[0]
                                           << "x" << maxGridSize[1]
                                           << "x" << maxGridSize[2]);
        // max threads per block
        int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
        DEBUG_PRINT("CUDA Max Threads Per Block: " << maxThreadsPerBlock);
        threadsPerBlock_ = 256;

        // max warp per block
        int maxWarpsPerBlock = deviceProp.maxThreadsPerBlock / deviceProp.warpSize;
        DEBUG_PRINT("CUDA Max Warps Per Block: " << maxWarpsPerBlock);

        // warp size
        int warpSize = deviceProp.warpSize;
        DEBUG_PRINT("CUDA Warp Size: " << warpSize);

        //
        DEBUG_PRINT("USING threadPerBlock: " << threadsPerBlock_);
        DEBUG_PRINT("USING workSize: " << workSize_);
        blocksPerGrid_ = (workSize_ + threadsPerBlock_ - 1) / threadsPerBlock_;
        DEBUG_PRINT("USING blockCount: " << blocksPerGrid_);
    }

    CudaWorker::~CudaWorker()
    {
        cleanup();
    }

    void CudaWorker::init()
    {
        DEBUG_PRINT("Initializing CudaWorker- allocate memory");
        // allocate memory
        hipMalloc(&u, (Nx_) * Ny_ * Nz * sizeof(double));
        hipMalloc(&u_0, (Nx_) * Ny_ * Nz * sizeof(double));
        hipMalloc(&v, Nx_ * (Ny_) * Nz * sizeof(double));
        hipMalloc(&v_0, Nx_ * (Ny_) * Nz * sizeof(double));
        hipMalloc(&w, Nx_ * Ny_ * (Nz) * sizeof(double));
        hipMalloc(&w_0, Nx_ * Ny_ * (Nz) * sizeof(double));

        hipMalloc(&avg_u, workSize_ * sizeof(double));
        hipMalloc(&avg_v, workSize_ * sizeof(double));
        hipMalloc(&avg_w, workSize_ * sizeof(double));

        hipMalloc(&omg_x, workSize_ * sizeof(double));
        hipMalloc(&omg_y, workSize_ * sizeof(double));
        hipMalloc(&omg_z, workSize_ * sizeof(double));

        hipMalloc(&f_x, workSize_ * sizeof(double));
        hipMalloc(&f_y, workSize_ * sizeof(double));
        hipMalloc(&f_z, workSize_ * sizeof(double));

        // allocate memory for density field
        hipMalloc(&density, workSize_ * sizeof(double));
        hipMalloc(&density_0, workSize_ * sizeof(double));

        // allocate memory for temperature field
        hipMalloc(&temperature, workSize_ * sizeof(double));
        hipMalloc(&temperature_0, workSize_ * sizeof(double));

        // allocate memory for transparency map
        hipMalloc(&transparency, workSize_ * sizeof(double));
    }

    void CudaWorker::cleanup()
    {
        DEBUG_PRINT("Cleaning up CudaWorker- free memory");
        // free memory
        hipFree(avg_u);
        hipFree(avg_v);
        hipFree(avg_w);
        hipFree(u);
        hipFree(v);
        hipFree(w);
        hipFree(u_0);
        hipFree(v_0);
        hipFree(w_0);

        hipFree(omg_x);
        hipFree(omg_y);
        hipFree(omg_z);

        hipFree(f_x);
        hipFree(f_y);
        hipFree(f_z);
    }

    void CudaWorker::copyDataToDevice(
        double *src, double *dst, int size)
    {
        hipMemcpy(dst, src, size * sizeof(double), hipMemcpyHostToDevice);
    }

    void CudaWorker::copyDataToHost(double *src, double *dst, int size)
    {
        hipMemcpy(dst, src, size * sizeof(double), hipMemcpyDeviceToHost);
    }

    void CudaWorker::setforceField(
        double *f_x,
        double *f_y,
        double *f_z)
    {
        copyDataToDevice(f_x, this->f_x, workSize_);
        copyDataToDevice(f_y, this->f_y, workSize_);
        copyDataToDevice(f_z, this->f_z, workSize_);
    }

    void CudaWorker::getforceField(
        double *f_x,
        double *f_y,
        double *f_z)
    {
        copyDataToHost(this->f_x, f_x, workSize_);
        copyDataToHost(this->f_y, f_y, workSize_);
        copyDataToHost(this->f_z, f_z, workSize_);
    }

    void CudaWorker::setVelocityField(
        double *u_src,
        double *v_src,
        double *w_src)
    {
        copyDataToDevice(u_src, this->u, (Nx_) * Ny_ * Nz_);
        copyDataToDevice(v_src, this->v, Nx_ * (Ny_) * Nz_);
        copyDataToDevice(w_src, this->w, Nx_ * Ny_ * (Nz_));
    }

    void CudaWorker::getVelocityField(
        double *u_dst,
        double *v_dst,
        double *w_dst)
    {
        copyDataToHost(this->u, u_dst, (Nx_) * Ny_ * Nz_);
        copyDataToHost(this->v, v_dst, Nx_ * (Ny_) * Nz_);
        copyDataToHost(this->w, w_dst, Nx_ * Ny_ * (Nz_));
    }

    void CudaWorker::getPreviosVelocityField(
        double *u_dst,
        double *v_dst,
        double *w_dst)
    {
        copyDataToHost(this->u_0, u_dst, (Nx_) * Ny_ * Nz_);
        copyDataToHost(this->v_0, v_dst, Nx_ * (Ny_) * Nz_);
        copyDataToHost(this->w_0, w_dst, Nx_ * Ny_ * (Nz_));
    }

    void CudaWorker::calculateVorticity()
    {
        // calculate average velocity
        // DEBUG_PRINT("Launching kernel");
        calculateAverageVelocityKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            u, v, w,
            avg_u, avg_v, avg_w,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();

        // calculate omg
        calculateOmgKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            avg_u, avg_v, avg_w,
            omg_x, omg_y, omg_z,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();

        // calculate vorticity force
        calculateVorticityForceKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            omg_x, omg_y, omg_z,
            f_x, f_y, f_z,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();
    }

    void CudaWorker::applyExternalForce()
    {
        applyExternalForceKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            u, v, w,
            f_x, f_y, f_z,
            workSize_, Nx_, Ny_, Nz_);
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            DEBUG_PRINT("CUDA Error: " << hipGetErrorString(error));
        }
        hipDeviceSynchronize();
    }

    void CudaWorker::advectVelocityField()
    {
        // copy current velocity field to previous
        hipMemcpy(u_0, u, (Nx_) * Ny_ * Nz_ * sizeof(double), hipMemcpyDeviceToDevice);
        hipMemcpy(v_0, v, Nx_ * (Ny_) * Nz_ * sizeof(double), hipMemcpyDeviceToDevice);
        hipMemcpy(w_0, w, Nx_ * Ny_ * (Nz_) * sizeof(double), hipMemcpyDeviceToDevice);

        advectVelocityFieldKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            u, v, w,
            u_0, v_0, w_0,
            workSize_, Nx_, Ny_, Nz_);
        // check error
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            DEBUG_PRINT("Error at advectVelocityFieldKernel: " << hipGetErrorString(error));
            DEBUG_PRINT("\tworkSize: " << workSize_);
            DEBUG_PRINT("\tNx: " << Nx_);
            DEBUG_PRINT("\tNy: " << Ny_);
            DEBUG_PRINT("\tNz: " << Nz_);
            DEBUG_PRINT("\tblocksPerGrid: " << blocksPerGrid_);
            DEBUG_PRINT("\tthreadsPerBlock: " << threadsPerBlock_);
        }
        hipDeviceSynchronize();
    }

    void CudaWorker::setDensityField(
        double *density)
    {
        // copy density field to device
        copyDataToDevice(density, this->density, workSize_);
    }

    void CudaWorker::getDensityField(
        double *density)
    {
        // copy density field to host
        copyDataToHost(this->density, density, workSize_);
    }

    void CudaWorker::setPreviosDensityField(
        double *density_0)
    {
        copyDataToDevice(density_0, this->density_0, workSize_);
    }

    void CudaWorker::getPreviosDensityField(
        double *density_0)
    {
        copyDataToHost(this->density_0,
                       density_0, workSize_);
    }

    void CudaWorker::setTemperatureField(
        double *temperature)
    {
        copyDataToDevice(temperature,
                         this->temperature, workSize_);
    }

    void CudaWorker::getTemperatureField(
        double *temperature)
    {
        copyDataToHost(this->temperature,
                       temperature, workSize_);
    }

    void CudaWorker::setPreviosTemperatureField(
        double *temperature_0)
    {
        copyDataToDevice(temperature_0,
                         this->temperature_0,
                         workSize_);
    }

    void CudaWorker::getPreviosTemperatureField(
        double *temperature_0)
    {
        copyDataToHost(this->temperature_0,
                       temperature_0, workSize_);
    }

    void CudaWorker::advectScalarField()
    {
        // copy velocity field to previous
        hipMemcpy(u_0, u,
                   (Nx_) * Ny_ * Nz_ * sizeof(double),
                   hipMemcpyDeviceToDevice);
        // copy current density field to previous
        hipMemcpy(density_0, density,
                   workSize_ * sizeof(double),
                   hipMemcpyDeviceToDevice);
        // copy current temperature field to previous
        hipMemcpy(temperature_0, temperature,
                   workSize_ * sizeof(double),
                   hipMemcpyDeviceToDevice);

        // advect density field
        advectScalarFieldKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            density, density_0,
            u_0, v_0, w_0,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();

        // advect temperature field
        advectScalarFieldKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            temperature, temperature_0,
            u_0, v_0, w_0,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();
    }

    __global__ void RayMarchingKernel(
        double *result,
        double *density,
        int Nx, int Ny, int Nz,
        double light_x, double light_y, double light_z,
        double module_scale_factor,
        double factor)
    {
        CUDA_FOR_EACH
        if (idx < Nx * Ny * Nz)
        {
            double sample_count = Nx;
            double step = 1.0;

            double pos_cell[3];
            getCenter<double>(i, j, k, pos_cell);

            double dir[3] = {
                light_x - pos_cell[0] / Nx * module_scale_factor,
                light_y - pos_cell[1] / Ny * module_scale_factor,
                light_z - pos_cell[2] / Nz * module_scale_factor};

            // normalize direction
            double norm = VEC3_NORM(dir[0], dir[1], dir[2]);
            dir[0] /= norm;
            dir[1] /= norm;
            dir[2] /= norm;

            double Tl = 1.0;
            for (int s = 0; s < sample_count; s++)
            {
                // update position
                pos_cell[0] += dir[0] * step;
                pos_cell[1] += dir[1] * step;
                pos_cell[2] += dir[2] * step;

                // if pos is out of bound
                if (pos_cell[0] > Nx ||
                    pos_cell[1] > Ny ||
                    pos_cell[2] > Nz)
                {
                    break;
                }

                // interpolate density
                double d = getScalar<double>(
                    pos_cell,
                    density,
                    Nx, Ny, Nz);

                if (d < 0.01)
                {
                    continue;
                }

                Tl *= exp(-factor * d * step / Nx);
                if (Tl < 0.01)
                {
                    break;
                }
            }
            result[idx] = Tl;
        }
    }

    void CudaWorker::genTransparencyMap(
        double light_x, double light_y, double light_z,
        double module_scale_factor,
        double factor)
    {
        RayMarchingKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            transparency,
            density,
            Nx_, Ny_, Nz_,
            light_x, light_y, light_z,
            module_scale_factor,
            factor);
        // check error
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            DEBUG_PRINT("Error at RayMarchingKernel: " << hipGetErrorString(error));
            DEBUG_PRINT("\tworkSize: " << workSize_);
            DEBUG_PRINT("\tNx: " << Nx_);
            DEBUG_PRINT("\tNy: " << Ny_);
            DEBUG_PRINT("\tNz: " << Nz_);
            DEBUG_PRINT("\tblocksPerGrid: " << blocksPerGrid_);
            DEBUG_PRINT("\tthreadsPerBlock: " << threadsPerBlock_);
        }

        hipDeviceSynchronize();
    }

    void CudaWorker::getTransparencyMap(
        double *transparency)
    {
        copyDataToHost(this->transparency, transparency, workSize_);
    }

}