#include "hip/hip_runtime.h"
#include "CudaWorker.h"
#include "common/debug.h"
#include "common/mmath.h"

#include <hip/hip_runtime.h>

#define VEC3_NORM(x, y, z) \
    sqrt((x) * (x) + (y) * (y) + (z) * (z))

#define VEC_CROSS(x1, y1, z1, x2, y2, z2, x, y, z) \
    x = y1 * z2 - z1 * y2;                         \
    y = z1 * x2 - x1 * z2;                         \
    z = x1 * y2 - y1 * x2;

namespace MCUDA
{
    //@todo might merge with the one in mmath.h
    template <typename T>
    __device__
        T
        cuda_linearInterpolation(
            const T *pt,
            T *src,
            int *dims,
            int *maxXYZ)
    {
        T pos[3];
        // clamp position
        pos[0] = min(
            max((T)0.0, pt[0]),
            (T)(maxXYZ[0]) * VOXEL_SIZE - (T)1e-6);
        pos[1] = min(
            max((T)0.0, pt[1]),
            (T)(maxXYZ[1]) * VOXEL_SIZE - (T)1e-6);
        pos[2] = min(
            max((T)0.0, pt[2]),
            (T)(maxXYZ[2]) * VOXEL_SIZE - (T)1e-6);

        int i = (int)(pos[0] / VOXEL_SIZE);
        int j = (int)(pos[1] / VOXEL_SIZE);
        int k = (int)(pos[2] / VOXEL_SIZE);

        T scale = 1.0 / VOXEL_SIZE;
        T fractx = scale * (pos[0] - i * VOXEL_SIZE);
        T fracty = scale * (pos[1] - j * VOXEL_SIZE);
        T fractz = scale * (pos[2] - k * VOXEL_SIZE);

        assert(fractx < 1.0 && fractx >= 0);
        assert(fracty < 1.0 && fracty >= 0);
        assert(fractz < 1.0 && fractz >= 0);

        // Y @ low X, low Z:
        T tmp1 = src[ACCESS3D(i, j, k)];
        T tmp2 = src[ACCESS3D(i, j + 1, k)];
        // Y @ high X, low Z:
        T tmp3 = src[ACCESS3D(i + 1, j, k)];
        T tmp4 = src[ACCESS3D(i + 1, j + 1, k)];

        // Y @ low X, high Z:
        T tmp5 = src[ACCESS3D(i, j, k + 1)];
        T tmp6 = src[ACCESS3D(i, j + 1, k + 1)];
        // Y @ high X, high Z:
        T tmp7 = src[ACCESS3D(i + 1, j, k + 1)];
        T tmp8 = src[ACCESS3D(i + 1, j + 1, k + 1)];

        // Y @ low X, low Z
        T tmp12 = ((T)(1) - fracty) * tmp1 + fracty * tmp2;
        // Y @ high X, low Z
        T tmp34 = ((T)(1) - fracty) * tmp3 + fracty * tmp4;

        // Y @ low X, high Z
        T tmp56 = ((T)(1) - fracty) * tmp5 + fracty * tmp6;
        // Y @ high X, high Z
        T tmp78 = ((T)(1) - fracty) * tmp7 + fracty * tmp8;

        // X @ low Z
        T tmp1234 = ((T)(1) - fractx) * tmp12 + fractx * tmp34;
        // X @ high Z
        T tmp5678 = ((T)(1) - fractx) * tmp56 + fractx * tmp78;

        // Z
        T tmp = ((T)(1) - fractz) * tmp1234 + fractz * tmp5678;
        return tmp;
    }

    __device__ double cuda_getVelocityX(
        double *pos_u, double *u, int Nx, int Ny, int Nz)
    {
        int dim[3] = {Nx + 1, Ny, Nz};
        int maxIndex[3] = {Nx, Ny - 1, Nz - 1};
        double pos_t[3];
        pos_t[0] = pos_u[0];
        pos_t[1] = pos_u[1] - 0.5 * VOXEL_SIZE;
        pos_t[2] = pos_u[2] - 0.5 * VOXEL_SIZE;
        return cuda_linearInterpolation<double>(
            pos_t,
            u,
            dim,
            maxIndex);
    }

    __device__ double cuda_getVelocityY(
        double *pos_v, double *v, int Nx, int Ny, int Nz)
    {
        int dim[3] = {Nx, Ny + 1, Nz};
        int maxIndex[3] = {Nx - 1, Ny, Nz - 1};
        double pos_t[3];
        pos_t[0] = pos_v[0] - 0.5 * VOXEL_SIZE;
        pos_t[1] = pos_v[1];
        pos_t[2] = pos_v[2] - 0.5 * VOXEL_SIZE;
        return cuda_linearInterpolation<double>(
            pos_t,
            v,
            dim,
            maxIndex);
    }

    __device__ double cuda_getVelocityZ(
        double *pos_w, double *w, int Nx, int Ny, int Nz)
    {
        int dim[3] = {Nx, Ny, Nz + 1};
        int maxIndex[3] = {Nx - 1, Ny - 1, Nz};
        double pos_t[3];
        pos_t[0] = pos_w[0] - 0.5 * VOXEL_SIZE;
        pos_t[1] = pos_w[1] - 0.5 * VOXEL_SIZE;
        pos_t[2] = pos_w[2];
        return cuda_linearInterpolation<double>(
            pos_t,
            w,
            dim,
            maxIndex);
    }

    __device__ double cuda_getVelocity(
        double *pos,
        double *vel,
        double *u,
        double *v,
        double *w,
        int Nx, int Ny, int Nz)
    {
        vel[0] = cuda_getVelocityX(pos, u, Nx, Ny, Nz);
        vel[1] = cuda_getVelocityY(pos, v, Nx, Ny, Nz);
        vel[2] = cuda_getVelocityZ(pos, w, Nx, Ny, Nz);
    }

    __global__ void calculateAverageVelocityKernel(
        double *u, double *v, double *w,
        double *avg_u, double *avg_v, double *avg_w,
        int workSize, int Nx, int Ny, int Nz)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        // calculate the index of the cell
        int i = idx % Nx;
        int j = (idx / Nx) % Ny;
        int k = idx / (Nx * Ny);

        if (idx < workSize)
        {
            // calculate average velocity
            avg_u[POS(i, j, k)] = (u[POS_X(i, j, k)] + u[POS_X(i + 1, j, k)]) * 0.5;
            avg_v[POS(i, j, k)] = (v[POS_Y(i, j, k)] + v[POS_Y(i, j + 1, k)]) * 0.5;
            avg_w[POS(i, j, k)] = (w[POS_Z(i, j, k)] + w[POS_Z(i, j, k + 1)]) * 0.5;
        }
    }

    __global__ void calculateOmgKernel(
        double *avg_u, double *avg_v, double *avg_w,
        double *omg_x, double *omg_y, double *omg_z,
        int workSize, int Nx, int Ny, int Nz)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        int i = idx % Nx;
        int j = (idx / Nx) % Ny;
        int k = idx / (Nx * Ny);

        if (idx < workSize)
        {
            // ignore boundary cells
            if (i == 0 || j == 0 || k == 0)
            {
                return;
            }
            if (i == Nx - 1 || j == Ny - 1 || k == Nz - 1)
            {
                return;
            }
            // calculate vorticity
            omg_x[POS(i, j, k)] = (avg_w[POS(i, j + 1, k)] - avg_w[POS(i, j - 1, k)] - avg_v[POS(i, j, k + 1)] + avg_v[POS(i, j, k - 1)]) * 0.5 / VOXEL_SIZE;
            omg_y[POS(i, j, k)] = (avg_u[POS(i, j, k + 1)] - avg_u[POS(i, j, k - 1)] - avg_w[POS(i + 1, j, k)] + avg_w[POS(i - 1, j, k)]) * 0.5 / VOXEL_SIZE;
            omg_z[POS(i, j, k)] = (avg_v[POS(i + 1, j, k)] - avg_v[POS(i - 1, j, k)] - avg_u[POS(i, j + 1, k)] + avg_u[POS(i, j - 1, k)]) * 0.5 / VOXEL_SIZE;
        }
    }

    __global__ void calculateVorticityForceKernel(
        double *omg_x, double *omg_y, double *omg_z,
        double *f_x, double *f_y, double *f_z,
        int workSize, int Nx, int Ny, int Nz)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        int i = idx % Nx;
        int j = (idx / Nx) % Ny;
        int k = idx / (Nx * Ny);

        if (idx < workSize)
        {
            // ignore boundary cells
            if (i == 0 || j == 0 || k == 0)
            {
                return;
            }
            if (i == Nx - 1 || j == Ny - 1 || k == Nz - 1)
            {
                return;
            }
            // compute gradient of vorticity
            double p, q;
            p = VEC3_NORM(omg_x[POS(i + 1, j, k)], omg_y[POS(i + 1, j, k)], omg_z[POS(i + 1, j, k)]);
            q = VEC3_NORM(omg_x[POS(i - 1, j, k)], omg_y[POS(i - 1, j, k)], omg_z[POS(i - 1, j, k)]);

            double grad1 = (p - q) / (2.0 * VOXEL_SIZE);

            p = VEC3_NORM(omg_x[POS(i, j + 1, k)], omg_y[POS(i, j + 1, k)], omg_z[POS(i, j + 1, k)]);
            q = VEC3_NORM(omg_x[POS(i, j - 1, k)], omg_y[POS(i, j - 1, k)], omg_z[POS(i, j - 1, k)]);
            double grad2 = (p - q) / (2.0 * VOXEL_SIZE);

            p = VEC3_NORM(omg_x[POS(i, j, k + 1)], omg_y[POS(i, j, k + 1)], omg_z[POS(i, j, k + 1)]);
            q = VEC3_NORM(omg_x[POS(i, j, k - 1)], omg_y[POS(i, j, k - 1)], omg_z[POS(i, j, k - 1)]);
            double grad3 = (p - q) / (2.0 * VOXEL_SIZE);

            double norm = VEC3_NORM(grad1, grad2, grad3);

            double ni = 0.0, nj = 0.0, nk = 0.0;
            if (norm != 0)
            {
                ni = grad1 / norm;
                nj = grad2 / norm;
                nk = grad3 / norm;
            }

            double f1, f2, f3;

            VEC_CROSS(
                omg_x[POS(i, j, k)],
                omg_y[POS(i, j, k)],
                omg_z[POS(i, j, k)],
                ni, nj, nk,
                f1, f2, f3);

            f_x[POS(i, j, k)] += VORT_EPS * VOXEL_SIZE * f1;
            f_y[POS(i, j, k)] += VORT_EPS * VOXEL_SIZE * f2;
            f_z[POS(i, j, k)] += VORT_EPS * VOXEL_SIZE * f3;
        }
    }

    __global__ void applyExternalForceKernel(
        double *u, double *v, double *w,
        double *f_x, double *f_y, double *f_z,
        int workSize, int Nx, int Ny, int Nz)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        int i = idx % Nx;
        int j = (idx / Nx) % Ny;
        int k = idx / (Nx * Ny);

        if (idx < workSize)
        {
            if (i < Nx - 1)
            {
                u[POS_X(i + 1, j, k)] += DT * (f_x[POS(i, j, k)] + f_x[POS(i + 1, j, k)]) * 0.5;
            }
            if (j < Ny - 1)
            {
                v[POS_Y(i, j + 1, k)] += DT * (f_y[POS(i, j, k)] + f_x[POS(i, j + 1, k)]) * 0.5;
            }
            if (k < Nz - 1)
            {
                w[POS_Z(i, j, k + 1)] += DT * (f_z[POS(i, j, k)] + f_x[POS(i, j, k + 1)]) * 0.5;
            }
        }
    }

    __global__ void advectVelocityFieldKernel(
        double *u, double *v, double *w,
        double *u_0, double *v_0, double *w_0,
        int workSize, int Nx, int Ny, int Nz)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int i = idx % Nx;
        int j = (idx / Nx) % Ny;
        int k = idx / (Nx * Ny);

        if (idx < workSize)
        {
            double half_dx = 0.5 * VOXEL_SIZE;

            double center[3];
            center[0] = half_dx + i * VOXEL_SIZE;
            center[1] = half_dx + j * VOXEL_SIZE;
            center[2] = half_dx + k * VOXEL_SIZE;

            double pos_u[3];
            pos_u[0] = center[0] - half_dx;
            pos_u[1] = center[1];
            pos_u[2] = center[2];

            double pos_v[3];
            pos_v[0] = center[0];
            pos_v[1] = center[1] - half_dx;
            pos_v[2] = center[2];

            double pos_w[3];
            pos_w[0] = center[0];
            pos_w[1] = center[1];
            pos_w[2] = center[2] - half_dx;

            // advect u

            double vel_u[3];
            cuda_getVelocity(
                pos_u,
                vel_u,
                u_0, v_0, w_0, Nx, Ny, Nz);

            double vel_v[3];
            cuda_getVelocity(
                pos_v,
                vel_v,
                u_0, v_0, w_0, Nx, Ny, Nz);

            double vel_w[3];
            cuda_getVelocity(
                pos_w,
                vel_w,
                u_0, v_0, w_0, Nx, Ny, Nz);

            pos_u[0] -= DT * vel_u[0];
            pos_u[1] -= DT * vel_u[1];
            pos_u[2] -= DT * vel_u[2];

            pos_v[0] -= DT * vel_v[0];
            pos_v[1] -= DT * vel_v[1];
            pos_v[2] -= DT * vel_v[2];

            pos_w[0] -= DT * vel_w[0];
            pos_w[1] -= DT * vel_w[1];
            pos_w[2] -= DT * vel_w[2];

            u[POS_X(i, j, k)] =
                cuda_getVelocityX(pos_u, u_0, Nx, Ny, Nz);

            v[POS_Y(i, j, k)] =
                cuda_getVelocityY(pos_v, v_0, Nx, Ny, Nz);

            w[POS_Z(i, j, k)] =
                cuda_getVelocityZ(pos_w, w_0, Nx, Ny, Nz);
        }
    }

    CudaWorker::CudaWorker(
        int workSize,
        int Nx,
        int Ny,
        int Nz) : workSize_(workSize), Nx_(Nx), Ny_(Ny), Nz_(Nz)
    {
        // check cuda device properties
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        if (deviceCount == 0)
        {
            DEBUG_PRINT("No CUDA devices found");
            exit(1);
        }
        DEBUG_PRINT("CUDA Device Count: " << deviceCount);
        hipSetDevice(0);

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        DEBUG_PRINT("CUDA Device Name: " << deviceProp.name);
        DEBUG_PRINT("CUDA Compute Capability: " << deviceProp.major << "." << deviceProp.minor);
        DEBUG_PRINT("CUDA Device Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << "MB");

        // SM count
        int SMCount = deviceProp.multiProcessorCount;
        DEBUG_PRINT("CUDA SM Count: " << SMCount);

        // max grid size
        int maxGridSize[3];
        maxGridSize[0] = deviceProp.maxGridSize[0];
        maxGridSize[1] = deviceProp.maxGridSize[1];
        maxGridSize[2] = deviceProp.maxGridSize[2];
        DEBUG_PRINT("CUDA Max Grid Size: " << maxGridSize[0]
                                           << "x" << maxGridSize[1]
                                           << "x" << maxGridSize[2]);
        // max threads per block
        int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
        DEBUG_PRINT("CUDA Max Threads Per Block: " << maxThreadsPerBlock);
        threadsPerBlock_ = maxThreadsPerBlock;

        // max warp per block
        int maxWarpsPerBlock = deviceProp.maxThreadsPerBlock / deviceProp.warpSize;
        DEBUG_PRINT("CUDA Max Warps Per Block: " << maxWarpsPerBlock);

        // warp size
        int warpSize = deviceProp.warpSize;
        DEBUG_PRINT("CUDA Warp Size: " << warpSize);

        //
        DEBUG_PRINT("USING threadPerBlock: " << threadsPerBlock_);
        DEBUG_PRINT("USING workSize: " << workSize_);
        blocksPerGrid_ = (workSize_ + threadsPerBlock_ - 1) / threadsPerBlock_;
        DEBUG_PRINT("USING blockCount: " << blocksPerGrid_);
    }

    CudaWorker::~CudaWorker()
    {
        cleanup();
    }

    void CudaWorker::init()
    {
        DEBUG_PRINT("Initializing CudaWorker- allocate memory");
        // allocate memory
        // why (Nx_ + 1) * Ny_ * Nz : because we need to store ,
        // an extra cell to call the boundary condition
        hipMalloc(&u, (Nx_ + 1) * Ny_ * Nz * sizeof(double));
        hipMalloc(&u_0, (Nx_ + 1) * Ny_ * Nz * sizeof(double));
        hipMalloc(&v, Nx_ * (Ny_ + 1) * Nz * sizeof(double));
        hipMalloc(&v_0, Nx_ * (Ny_ + 1) * Nz * sizeof(double));
        hipMalloc(&w, Nx_ * Ny_ * (Nz + 1) * sizeof(double));
        hipMalloc(&w_0, Nx_ * Ny_ * (Nz + 1) * sizeof(double));

        hipMalloc(&avg_u, workSize_ * sizeof(double));
        hipMalloc(&avg_v, workSize_ * sizeof(double));
        hipMalloc(&avg_w, workSize_ * sizeof(double));

        hipMalloc(&omg_x, workSize_ * sizeof(double));
        hipMalloc(&omg_y, workSize_ * sizeof(double));
        hipMalloc(&omg_z, workSize_ * sizeof(double));

        hipMalloc(&f_x, workSize_ * sizeof(double));
        hipMalloc(&f_y, workSize_ * sizeof(double));
        hipMalloc(&f_z, workSize_ * sizeof(double));
    }

    void CudaWorker::cleanup()
    {
        DEBUG_PRINT("Cleaning up CudaWorker- free memory");
        // free memory
        hipFree(avg_u);
        hipFree(avg_v);
        hipFree(avg_w);

        hipFree(omg_x);
        hipFree(omg_y);
        hipFree(omg_z);

        hipFree(f_x);
        hipFree(f_y);
        hipFree(f_z);
    }

    void CudaWorker::copyDataToDevice(
        double *src, double *dst, int size)
    {
        hipMemcpy(dst, src, size * sizeof(double), hipMemcpyHostToDevice);
    }

    void CudaWorker::copyDataToHost(double *src, double *dst, int size)
    {
        hipMemcpy(dst, src, size * sizeof(double), hipMemcpyDeviceToHost);
    }

    void CudaWorker::setforceField(
        double *f_x,
        double *f_y,
        double *f_z)
    {
        copyDataToDevice(f_x, this->f_x, workSize_);
        copyDataToDevice(f_y, this->f_y, workSize_);
        copyDataToDevice(f_z, this->f_z, workSize_);
    }

    void CudaWorker::getforceField(
        double *f_x,
        double *f_y,
        double *f_z)
    {
        copyDataToHost(this->f_x, f_x, workSize_);
        copyDataToHost(this->f_y, f_y, workSize_);
        copyDataToHost(this->f_z, f_z, workSize_);
    }

    void CudaWorker::setVelocityField(
        double *u,
        double *v,
        double *w)
    {
        copyDataToDevice(u, this->u, (Nx_ + 1) * Ny_ * Nz_);
        copyDataToDevice(v, this->v, Nx_ * (Ny_ + 1) * Nz_);
        copyDataToDevice(w, this->w, Nx_ * Ny_ * (Nz_ + 1));
    }

    void CudaWorker::getVelocityField(
        double *u,
        double *v,
        double *w)
    {
        copyDataToHost(this->u, u, (Nx_ + 1) * Ny_ * Nz_);
        copyDataToHost(this->v, v, Nx_ * (Ny_ + 1) * Nz_);
        copyDataToHost(this->w, w, Nx_ * Ny_ * (Nz_ + 1));
    }

    void CudaWorker::calculateVorticity()
    {
        // calculate average velocity
        // DEBUG_PRINT("Launching kernel");
        calculateAverageVelocityKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            u, v, w,
            avg_u, avg_v, avg_w,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();

        // calculate omg
        calculateOmgKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            avg_u, avg_v, avg_w,
            omg_x, omg_y, omg_z,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();

        // calculate vorticity force
        calculateVorticityForceKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            omg_x, omg_y, omg_z,
            f_x, f_y, f_z,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();
    }

    void CudaWorker::applyExternalForce()
    {
        applyExternalForceKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            u, v, w,
            f_x, f_y, f_z,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();
    }

    void CudaWorker::advectVelocityField()
    {
        // swap u and u_0
        double *temp = u;
        u = u_0;
        u_0 = temp;

        // swap v and v_0
        temp = v;
        v = v_0;
        v_0 = temp;

        // swap w and w_0
        temp = w;
        w = w_0;
        w_0 = temp;

        advectVelocityFieldKernel<<<blocksPerGrid_, threadsPerBlock_>>>(
            u, v, w,
            u_0, v_0, w_0,
            workSize_, Nx_, Ny_, Nz_);
        hipDeviceSynchronize();
    }

}

__global__ void addKernel()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
}

void test_cuda()
{
    // launch kernel
    DEBUG_PRINT("Launching kernel\n");
    addKernel<<<4, 256>>>();
    hipDeviceSynchronize();
    DEBUG_PRINT("Kernel finished\n");
}